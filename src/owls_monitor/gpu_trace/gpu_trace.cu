#include "hip/hip_runtime.h"
#include <algorithm>
#include <assert.h>
#include <cstddef>
#include <cstdint>
#include <cstdlib>
#include <cstring>
#include <exception>
#include <execinfo.h>
#include <fstream>
#include <functional>
#include <iostream>
#include <iterator>
#include <map>
#include <memory>
#include <ostream>
#include <sstream>
#include <stdint.h>
#include <stdio.h>
#include <string>
#include <sys/stat.h>
#include <unistd.h>
#include <unordered_map>
#include <unordered_set>
#include <utility>
#include <vector>

#include "helper/json.hpp"
#include "instr_types.h"
#include "nvbit.h"
#include "nvbit_tool.h"

#include "utils/channel.hpp"

#include "bt.hpp"
#include "cfg.hpp"
#include "dump.hpp"
#include "helper/common.h"
#include "helper/debug.h"
#include "helper/dump.h"
#include "helper/pipe/pipe.h"
#include "helper/shm.hpp"

using namespace std;

#define HEX(x) "0x" << setfill('0') << setw(16) << hex << (u64)x << dec

#define CHANNEL_SIZE (1l << 20)

#define KERNEL_INFO_FILE "kernel.info"

struct CTXstate {

  int id;

  ChannelDev *channel_dev;
  ChannelHost channel_host;
};

pthread_mutex_t mutex;

unordered_map<hipCtx_t, CTXstate *> ctx_state_map;

SharedMemory<AllocBasicInfo> sharedMemory(1024);

CTXstate *get_ctx_state(hipCtx_t ctx) {
  assert(ctx_state_map.find(ctx) != ctx_state_map.end());
  return ctx_state_map[ctx];
}

void set_ctx_state(hipCtx_t ctx, CTXstate *ctx_state) {
  assert(ctx_state_map.find(ctx) == ctx_state_map.end());
  ctx_state_map[ctx] = ctx_state;
}

bool skip_callback_flag = false;

int verbose = 0;
int debug = 0;
#define DEBUG(x) if (debug >= x)

u32 kernel_launch_id = 1;

bool doesRun = true;

map<string, vector<u64>> f_bb;
map<string, u64> f_id;

unordered_set<hipFunction_t> already_instrumented;
unordered_set<hipCtx_t> already_send_kernel_start;

map<KernelID, shared_ptr<KernelContext>> kernel_info;

void dump_address(u8 *mem, u32 length) {
  DEBUG(3) {
    WARNF("dumping memory - start address : %p", mem);
    for (u32 i = 0; i < length; i++) {
      printf("0x%02x ", mem[i]);
    }
    printf("\n");
  }
}

bool create_dir(const std::string &path) {
  struct stat info;
  if (stat(path.c_str(), &info) != 0) {
    if (mkdir(path.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) != 0) {

      return false;
    }
  } else if (!(info.st_mode & S_IFDIR)) {

    return false;
  }
  return true;
}


void send_context_start() {
  if (verbose)
    ACTF("Context Start");
}

void send_context_stop() {
  if (verbose)
    ACTF("Context Stop");
}

void send_kernel_start_data(u32 kernel_id, u64 func_id, Dim block, Dim thread) {

  if (verbose)
    ACTF("Kernel start : 0x%lx - %d", func_id, kernel_id);
}

void call_owls_device_trace(Instr *instr, trace_args_t ta) {
  nvbit_insert_call(instr, "__owls_device_trace", IPOINT_BEFORE);
  nvbit_add_call_arg_guard_pred_val(instr);
  nvbit_add_call_arg_const_val32(instr, ta.type);
  nvbit_add_call_arg_const_val64(instr, ta.bb_id);
  nvbit_add_call_arg_const_val64(instr, ta.desc);

  switch (ta.type) {
  case TRACE_MEM_LOAD:
  case TRACE_MEM_STORE: {
    nvbit_add_call_arg_mref_addr64(instr, ta.extra);
    break;
  }
  case TRACE_FUNC_CALL:
  case TRACE_BB_ACCESS:
  case TRACE_FUNC_RET: {
    nvbit_add_call_arg_const_val64(instr, ta.extra);
    break;
  }
  }
  nvbit_add_call_arg_const_val64(instr, ta.pchannel_dev);
}

void instrument_mem_access(hipCtx_t ctx, basic_block_t *bb, u64 bb_id) {
  auto get_instr_type = [](Instr *instr) {
    u8 type = TRACE_UNKNOWN;
    if (instr->isLoad())
      type = TRACE_MEM_LOAD;
    else if (instr->isStore())
      type = TRACE_MEM_STORE;
    return type;
  };

  for (auto instr : bb->instrs) {
    if (instr->getMemorySpace() == InstrType::MemorySpace::NONE ||
        instr->getMemorySpace() == InstrType::MemorySpace::CONSTANT) {
      continue;
    }

    unsigned long mref_idx = 0;
    for (int i = 0; i < instr->getNumOperands(); i++) {
      const InstrType::operand_t *op = instr->getOperand(i);

      if (op->type == InstrType::OperandType::MREF) {
        u8 type = get_instr_type(instr);
        if (type == TRACE_UNKNOWN)
          continue;

        trace_args_t ta = {
            .type = type,
            .bb_id = bb_id,
            .desc = (u64)instr->getMemorySpace() << 48 |
                    (u64)instr->getSize() << 32 | (u32)instr->getOffset(),
            .extra = mref_idx,
            .pchannel_dev = (u64)get_ctx_state(ctx)->channel_dev,
        };
        call_owls_device_trace(instr, ta);
        DEBUG(3)
        WARNF("Memory Access in Instr %d", instr->getIdx());
        mref_idx++;
      }
    }
  }
}

unordered_set<u32> func_entries;
void collect_func_entry(hipCtx_t ctx, hipFunction_t f) {
  func_entries.clear();
  func_entries.emplace(0);
  for (auto instr : nvbit_get_instrs(ctx, f)) {
    if (strcmp(instr->getOpcodeShort(), "CALL") == 0) {
      for (int i = 0; i < instr->getNumOperands(); i++) {
        const InstrType::operand_t *op = instr->getOperand(i);
        auto func_entry = op->u.imm_uint64.value;
        func_entries.emplace(func_entry);
      }
    }
  }
}

void instrument_bb_access(hipCtx_t ctx, basic_block_t *bb, u64 bb_id) {

  auto first_instr = bb->instrs[0];
  auto offset = first_instr->getOffset();

  u8 type = TRACE_BB_ACCESS;

  trace_args_t ta = {
      .type = type,
      .bb_id = bb_id,
      .desc = bb->instrs.size(),
      .pchannel_dev = (u64)get_ctx_state(ctx)->channel_dev,
  };
  call_owls_device_trace(first_instr, ta);

}

void instrument_basic_block(hipCtx_t ctx, basic_block_t *bb, u64 bb_id) {
  DEBUG(3) {
    WARNF("instrument_basic_block in %p , id : 0x%lx", bb, bb_id);
    for (auto i : bb->instrs) {
      i->print();
    }
  }
  instrument_bb_access(ctx, bb, bb_id);
  instrument_mem_access(ctx, bb, bb_id);
}

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
  auto related_functions = nvbit_get_related_functions(ctx, func);
  related_functions.push_back(func);

  for (auto f : related_functions) {
    auto func_name = nvbit_get_func_name(ctx, f);

    if (verbose)
      ACTF("instrument_function in function : %s", func_name);

    auto cfg = nvbit_get_CFG(ctx, f);

    string src;
    hash<string> h;

    u64 func_id =
        (f_id.find(func_name) != f_id.end()) ? f_id[func_name] : h(func_name);
    f_id[func_name] = func_id;

    vector<u64> bb_ids;
    for (auto bb : cfg.bbs) {

      u64 bb_id =
          ((u64)kernel_launch_id << 32) | (u32)(bb->instrs[0]->getOffset());

      bb_ids.push_back(bb_id);
      instrument_basic_block(ctx, bb, bb_id);

      /*


      char *fname, *dname; u32 line, line_stop;
      nvbit_get_line_info(ctx, f, bb->instrs[0]->getOffset(), &fname, &dname,
      &line); nvbit_get_line_info(ctx, f,
      bb->instrs[bb->instrs.size()-1]->getOffset(), &fname, &dname, &line_stop);
      bool is_valid_str = true;
      for (auto dname_c: string(dname)) {
        is_valid_str &= isprint(dname_c);
      }
      for (auto fname_c: string(fname)) {
        is_valid_str &= isprint(fname_c);
      }
      if (is_valid_str && verbose)
        ACTF("0x%lx: %s/%s:%u-%u",bb_id, dname, fname, line, line_stop);
      */
    }
    f_bb[func_name] = bb_ids;
  }
}

__global__ void flush_channel(ChannelDev *ch_dev) { ch_dev->flush(); }

BackTrace get_backtrace(size_t max_size = 100) {
  BackTrace bt;
  void *array[max_size];
  char **strings;
  int size;

  size = backtrace(array, max_size);
  if (size <= 0)
    return bt;

  strings = backtrace_symbols(array, size);
  if (strings != NULL) {

    for (int i = 6; i < size; i++) {

      bt.push_back(owls::extract_func_info(strings[i]));
    }
  }

  free(strings);

  return bt;
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
  pthread_mutex_lock(&mutex);

  /* we prevent re-entry on this callback when issuing CUDA functions inside
   * this function */
  if (skip_callback_flag) {
    pthread_mutex_unlock(&mutex);
    return;
  }
  skip_callback_flag = true;

  if (cbid == API_CUDA_cuLaunchKernel_ptsz || cbid == API_CUDA_cuLaunchKernel) {
    cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

    hipDeviceSynchronize();
    assert(hipGetLastError() == hipSuccess);

    if (!is_exit) {

      instrument_function_if_needed(ctx, p->f);

      int nregs = 0;
      CUDA_SAFECALL(
          hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

      int shmem_static_nbytes = 0;
      CUDA_SAFECALL(hipFuncGetAttribute(
          &shmem_static_nbytes, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));

      const char *func_name = nvbit_get_func_name(ctx, p->f);
      u64 pc = nvbit_get_func_addr(p->f);

      nvbit_set_at_launch(ctx, p->f, &kernel_launch_id, sizeof(u32));

      doesRun = true;

      auto block = Dim{p->gridDimX, p->gridDimY, p->gridDimZ};
      auto thread = Dim{p->blockDimX, p->blockDimY, p->blockDimZ};

      MemPool abi = sharedMemory.parse();

      kernel_info[kernel_launch_id] =
          make_shared<KernelContext>(KernelContext{.funcName = func_name,
                                                   .kernelID = kernel_launch_id,
                                                   .funcID = f_id[func_name],
                                                   .memPool = abi,
                                                   .builder = DCFGBuilder(),
                                                   .bt = get_backtrace()});
      send_kernel_start_data(kernel_launch_id++, f_id[func_name], block,
                             thread);

      bool flag = !getenv("NOINSTR");
      nvbit_enable_instrumented(ctx, p->f, flag);

      OKF("CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - Kernel "
          "name %s - grid launch id %ld - grid size %d,%d,%d - block "
          "size %d,%d,%d - nregs %d - shmem %d - cuda stream id %ld",
          (u64)ctx, pc, func_name, kernel_launch_id, p->gridDimX, p->gridDimY,
          p->gridDimZ, p->blockDimX, p->blockDimY, p->blockDimZ, nregs,
          shmem_static_nbytes + p->sharedMemBytes, (u64)p->hStream);
    } else {
      nvbit_enable_instrumented(ctx, p->f, false);
    }
  }
  skip_callback_flag = false;
  pthread_mutex_unlock(&mutex);
}

void deserialize_trace(u8 *buffer, u32 len) {
  u32 offset = 0;
  u8 *cur_buffer;
  u8 type;
  while (offset < len) {
    cur_buffer = buffer + offset;

    if (!(*cur_buffer & TYPE_GPU))
      type = 0xff;
    else
      type = *cur_buffer & ~TYPE_GPU;

    switch (type) {

    case TRACE_BB_ACCESS: {
      bb_record_t *r = (bb_record_t *)cur_buffer;

      kernel_info[r->info.kernelid]->builder.record_bb_access(
          r->info.ctaid, r->info.warpid, r->info.bbid);

      offset += sizeof(bb_record_t);
      break;
    }

    case TRACE_MEM_LOAD:
    case TRACE_MEM_STORE:
    case TRACE_MEM_ATOMIC: {
      mem_record_t *r = (mem_record_t *)cur_buffer;

      auto bma = make_shared<BasicMemAccess>();

      for (int i = 0; i < 32; i++) {
        if (!((r->info.active >> i) % 2))
          continue;
        auto addr = r->addr[i];
        (*bma)[addr] =
            ((*bma).find(addr) != (*bma).end()) ? (*bma)[addr] + 1 : 1;
      }

      MemAccess ma;
      ma[InstrType::MemorySpaceStr[r->type]] = bma;

      try {

        kernel_info[r->info.kernelid]->builder.record_mem_access(
            r->info.ctaid, r->info.warpid, r->info.bbid, r->offset, ma);
      } catch (const char *&e) {
        auto funcID = kernel_info[r->info.kernelid]->funcID;
        for (auto f : f_id) {
          if (f.second == funcID) {
            auto funcName = f.first;
            BADF("record mem access failed. %s", e);
            cerr << "func name : " << funcName << endl;
            break;
          }
        }

        exit(EXIT_FAILURE);
      }

      offset += sizeof(mem_record_t);
      break;
    }
    default:
      BADF("deserialize error! unsupported type : %s", getTraceTypeStr(type));
    }
  }
}

void *recv_thread_fun(void *args) {
  hipCtx_t ctx = (hipCtx_t)args;

  pthread_mutex_lock(&mutex);

  CTXstate *ctx_state = get_ctx_state(ctx);

  ChannelHost *ch_host = &ctx_state->channel_host;
  pthread_mutex_unlock(&mutex);
  char *recv_buffer = (char *)malloc(CHANNEL_SIZE);

  while (true) {
    u32 num_recv_bytes = ch_host->recv(recv_buffer, CHANNEL_SIZE);
    if (num_recv_bytes > 0) {

      deserialize_trace((u8 *)recv_buffer, num_recv_bytes);
    } else if (!doesRun) {
      break;
    }
  }
  free(recv_buffer);
  return NULL;
}

void DumpJson(nlohmann::json j) {
  if (char *dirname = getenv("OWL_TRACE")) {
    create_dir(dirname);

    char *kernel_filename =
        new char[strlen(dirname) + strlen("/kernel.json") + 1];
    strcpy(kernel_filename, dirname);
    strcat(kernel_filename, "/kernel.json");
    owls::json_to_file(j, kernel_filename);
  }
  if (char *pipename = getenv("OWLS_PIPE2"))
    owls::json_to_pipe(j, pipename);
  if (char *filename = getenv("OWLS_FILE"))
    owls::json_to_file(j, filename);
}

void collect_kernel_trace() {
  DEBUG(1) ACTF("staring collect kernel trace");
  cout << flush;
  vector<KernelTrace> kernel_traces;
  int idx = 0;
  for (auto iter = kernel_info.begin(); iter != kernel_info.end();) {
    auto trace = iter->second->collect();
    kernel_traces.push_back(trace);

    if (verbose)
      ACTF("collect kernel 0x%lx-%lu", trace.funcID, trace.kernelID);

    iter->second.reset();
    kernel_info.erase(iter++);
  }

  DEBUG(1) ACTF("staring dump kernel trace");
  nlohmann::json j = owls::to_json(kernel_traces);

  DumpJson(j);
}

void nvbit_at_init() {
  setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
  GET_VAR_INT(verbose, "VERBOSE", 0, "Enable verbosity inside the tool");
  GET_VAR_INT(debug, "DEBUG", 0, "Enable debug info inside the tool");
  if (!getenv("NOBANNER")) {
    string pad(100, '-');
    printf("%s\n", pad.c_str());
  }

  pthread_mutexattr_t attr;
  pthread_mutexattr_init(&attr);
  pthread_mutexattr_settype(&attr, PTHREAD_MUTEX_RECURSIVE);
  pthread_mutex_init(&mutex, &attr);
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
  pthread_mutex_lock(&mutex);
  OKF("STARTING CONTEXT %p", ctx);

  if (!sharedMemory.attach()) {
    std::cerr << "Failed to attach shared memory!" << std::endl;
    return;
  }

  send_context_start();
  CTXstate *ctx_state = new CTXstate;

  set_ctx_state(ctx, ctx_state);

  hipMallocManaged(&ctx_state->channel_dev, sizeof(ChannelDev));
  ctx_state->channel_host.init((int)ctx_state_map.size() - 1, CHANNEL_SIZE,
                               ctx_state->channel_dev, recv_thread_fun, ctx);
  nvbit_set_tool_pthread(ctx_state->channel_host.get_thread());
  pthread_mutex_unlock(&mutex);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
  pthread_mutex_lock(&mutex);
  skip_callback_flag = true;

  CTXstate *ctx_state = get_ctx_state(ctx);

  flush_channel<<<1, 1>>>(ctx_state->channel_dev);

  hipDeviceSynchronize();
  doesRun = false;
  assert(hipGetLastError() == hipSuccess);

  ofstream f_info;

  ctx_state->channel_host.destroy(false);
  hipFree(ctx_state->channel_dev);
  skip_callback_flag = false;
  delete ctx_state;

  sharedMemory.detach();

  send_context_stop();
  collect_kernel_trace();
  OKF("TERMINATING CONTEXT %p", ctx);

  pthread_mutex_unlock(&mutex);
}
