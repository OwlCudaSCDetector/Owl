#include "hip/hip_runtime.h"

#include <cstdint>
#include <cstdio>
#include <cstdlib>

#include <stdint.h>
#include <stdio.h>

#include "utils/utils.h"

#include "utils/channel.hpp"

#include "helper/common.h"

__device__ void dump_device_address(u8 *mem, u32 length) {
  printf("[device] dumping memory - start address : %p\n", mem);
  for (u32 i = 0; i < length; i++) {
    printf("0x%02x ", mem[i]);
  }
  printf("\n");
}

__device__ void dump_common_info(common_info_t &common_info) {
  printf(
      "[device] common_info : type:%d, smid:%d, warpid:%d, ctaid:{%d,%d,%d}\n",
      common_info.type, common_info.smid, common_info.warpid,
      common_info.ctaid.x, common_info.ctaid.y, common_info.ctaid.z);
}

extern "C" __device__ __noinline__ void __owl_device_trace(int pred, u32 type,
                                                            u64 bb_id, u64 desc,
                                                            u64 extra,
                                                            u64 pchannel_dev) {

  if (!pred) {
    return;
  }

  u32 lane_id = get_laneid();

  u32 active = __activemask();
  u32 active_mask = __ballot_sync(__activemask(), 1);
  u32 lowest = __ffs(active) - 1;

  u32 length = sizeof(common_info_t);
  u8 *cur_record = nullptr;

  common_info_t common_info;
  common_info.type = GPU_MODE((u8)type);
  common_info.smid = get_smid();
  common_info.warpid = get_warpid();

  common_info.ctaid.x = blockIdx.x;
  common_info.ctaid.y = blockIdx.y;
  common_info.ctaid.z = blockIdx.z;
  common_info.active = active;
  common_info.kernelid = u32(bb_id >> 32);
  common_info.bbid = bb_id & 0xffffffff;

  mem_record_t mem_record;
  bb_record_t bb_record;

  switch (type) {
  case TRACE_MEM_LOAD:
  case TRACE_MEM_STORE:
  case TRACE_MEM_ATOMIC: {
    mem_record.info = common_info;
    mem_record.type = (desc >> 48) & 0xffffffff;
    mem_record.size = (desc >> 32) & 0xffffffff;
    mem_record.offset = desc & 0xffffffff;

    for (u8 i = 0; i < warpSize; i++) {
      mem_record.addr[i] = __shfl_sync(active_mask, extra, i);
    }
    length = sizeof(mem_record_t);
    cur_record = (u8 *)&mem_record;
    break;
  }

  case TRACE_FUNC_CALL:
  case TRACE_FUNC_RET:
  case TRACE_BB_ACCESS: {
    bb_record.info = common_info;
    bb_record.instr_num = desc;
    length = sizeof(bb_record_t);
    cur_record = (u8 *)&bb_record;
    break;
  }
  }

  if (lane_id == lowest) {

    ChannelDev *channel_dev = (ChannelDev *)pchannel_dev;

    channel_dev->push(cur_record, length);
  }
}