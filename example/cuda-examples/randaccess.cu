
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>
#include <iostream>

__global__ void randomAccessKernel(int **array, int rows, int cols,
                                   int randValue, int randValue2) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int row = (threadIdx.x * 2 + tid + randValue) % rows;
  int col = ((threadIdx.x + 1) * 3 + tid + randValue2) % cols;

  if (row < rows && col < cols) {
    array[row][col] = tid;
  }
}

int main(int argc, char *argv[]) {
  int rows = 100;
  int cols = 100;
  int numThreads = 1024;

  if (argc > 1) {
    numThreads = std::atoi(argv[1]);
  }

  std::srand(std::time(0));

  int **array;
  hipMallocManaged(&array, rows * sizeof(int *));
  for (int i = 0; i < rows; i++) {
    hipMallocManaged(&array[i], cols * sizeof(int));
  }

  int numBlocks = (numThreads + 255) / 256;
  dim3 gridDim(numBlocks, 1, 1);
  dim3 blockDim(256, 1, 1);

  int randValue = std::rand();
  int randValue2 = std::rand();
  std::cout << randValue << " " << randValue2 << std::endl;

  randomAccessKernel<<<gridDim, blockDim>>>(array, rows, cols, randValue,
                                            randValue2);
  hipDeviceSynchronize();

  for (int i = 0; i < rows; i++) {
    hipFree(array[i]);
  }
  hipFree(array);

  return 0;
}
