#include "hip/hip_runtime.h"
#include "sha1.hh"

#include <stdint.h>
#include <assert.h>

__device__ uint32_t swap(uint32_t v)
{
	return ((v & 0x000000ffU) << 24) |
	       ((v & 0x0000ff00U) << 8) |
	       ((v & 0x00ff0000U) >> 8) |
	       ((v & 0xff000000U) >> 24);
}

typedef struct hash_digest
{
	uint32_t h1;
	uint32_t h2;
	uint32_t h3;
	uint32_t h4;
	uint32_t h5;
} hash_digest_t;

#define HMAC

__inline__ __device__ void getBlock(char* buf, int offset, int len, uint32_t* dest)
{
	uint32_t *tmp;

	unsigned int tempbuf[16];

	tmp = (uint32_t*)(buf + offset);
	if (offset + 64 <= len) {
		#pragma unroll 16
		for (unsigned i = 0; i < 16; i++) {
			dest[i] = swap(tmp[i]);
		}
	} else if (len > offset && (len - offset) < 56) { //case 1 enough space in last block for padding
		unsigned i;
		for (i = 0; i < (len - offset) / 4; i++) {
			tempbuf[i] = swap(tmp[i]);
		}

		switch (len % 4) {
		case 0:
			tempbuf[i] = swap(0x00000080);
			i++;
			break;
		case 1:
			tempbuf[i] = swap(0x00008000 | (tmp[i] & 0x000000FF));
			i++;
			break;
		case 2:
			tempbuf[i] = swap(0x00800000 | (tmp[i] & 0x0000FFFF));
			i++;
			break;
		case 3:
			tempbuf[i] = swap(0x80000000 | (tmp[i] & 0x00FFFFFF));
			i++;
			break;
			};
		for ( ; i < 14; i++) {
			tempbuf[i] = 0;
		}
		#pragma unroll 14
		for (i = 0; i < 14; i++) {
			dest[i] = tempbuf[i];
		}
		dest[14] = 0x00000000;
		#ifndef HMAC
		dest[15] = len * 8;
		#else
		dest[15] = (len + 64) * 8;
		#endif

	} else if (len > offset && (len - offset) >= 56 ) {
		//case 2 not enough space in last block (containing message) for padding
	        unsigned i;
		for (i = 0; i < (len - offset)/4 ; i++ ) {
			tempbuf[i] = swap(tmp[i]);
		}
		switch (len % 4) {
		case 0:
			tempbuf[i] = swap(0x00000080);
			i++;
			break;
		case 1:
			tempbuf[i] = swap(0x00008000 | (tmp[i] & 0x000000FF));
			i++;
			break;
		case 2:
			tempbuf[i] = swap(0x00800000 | (tmp[i] & 0x0000FFFF));
			i++;
			break;
		case 3:
			tempbuf[i] = swap(0x80000000 | (tmp[i] & 0x00FFFFFF));
			i++;
			break;
		};

		for ( ; i < 16; i++){
			tempbuf[i] = 0x00000000;
		}

		#pragma unroll 16
		for (i = 0; i < 16; i++) {
			dest[i] = tempbuf[i];
		}

	} else if (offset == len) { //message end is aligned in 64 bytes
		dest[0] = swap(0x00000080);
		#pragma unroll 13
		for (unsigned i = 1; i < 14; i++)
			dest[i] = 0x00000000;
		dest[14] = 0x00000000;
		#ifndef HMAC
		dest[15] = len * 8;
		#else
		dest[15] = (len + 64) * 8;
		#endif

	} else if (offset  >  len) { //the last block in case 2
		#pragma unroll 14
		for (unsigned i = 0; i < 14; i++)
			dest[i] = 0x00000000;
		dest[14] = 0x00000000;
		#ifndef HMAC
		dest[15] = len * 8;
		#else
		dest[15] = (len + 64) * 8;
		#endif

	} else {
	}
}


__device__ void computeSHA1Block(char* in, uint32_t* w, int offset, int len,  hash_digest_t &h)
{
	uint32_t a = h.h1;
	uint32_t b = h.h2;
	uint32_t c = h.h3;
	uint32_t d = h.h4;
	uint32_t e = h.h5;
	uint32_t f;
	uint32_t k;
	uint32_t temp;

	getBlock(in, offset, len, w);

	k = 0x5A827999;
	//0 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
	w[0] = w[0] << 1 | w[0] >> 31;

	//1 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
	w[1] = w[1] << 1 | w[1] >> 31;

	//2 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
	w[2] = w[2] << 1 | w[2] >> 31;

	//3 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
	w[3] = w[3] << 1 | w[3] >> 31;

	//4 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
	w[4] = w[4] << 1 | w[4] >> 31;

	//5 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
	w[5] = w[5] << 1 | w[5] >> 31;

	//6 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
	w[6] = w[6] << 1 | w[6] >> 31;

	//7 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
	w[7] = w[7] << 1 | w[7] >> 31;

	//8 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
	w[8] = w[8] << 1 | w[8] >> 31;

	//9 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
	w[9] = w[9] << 1 | w[9] >> 31;

	//10 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
	w[10] = w[10] << 1 | w[10] >> 31;

	//11 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
	w[11] = w[11] << 1 | w[11] >> 31;

	//12 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
	w[12] = w[12] << 1 | w[12] >> 31;

	//13 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
	w[13] = w[13] << 1 | w[13] >> 31;

	//14 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
	w[14] = w[14] << 1 | w[14] >> 31;

	//15 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
	w[15] = w[15] << 1 | w[15] >> 31;

	//16 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
	w[0] = w[0] << 1 | w[0] >> 31;

	//17 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
	w[1] = w[1] << 1 | w[1] >> 31;

	//18 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
	w[2] = w[2] << 1 | w[2] >> 31;

	//19 of 0-20
	f = (b & c) | ((~b) & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
	w[3] = w[3] << 1 | w[3] >> 31;

	k = 0x6ED9EBA1;
	//20 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
	w[4] = w[4] << 1 | w[4] >> 31;

	//21 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
	w[5] = w[5] << 1 | w[5] >> 31;

	//22 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
	w[6] = w[6] << 1 | w[6] >> 31;

	//23 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
	w[7] = w[7] << 1 | w[7] >> 31;

	//24 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
	w[8] = w[8] << 1 | w[8] >> 31;

	//25 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
	w[9] = w[9] << 1 | w[9] >> 31;

	//26 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
	w[10] = w[10] << 1 | w[10] >> 31;

	//27 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
	w[11] = w[11] << 1 | w[11] >> 31;

	//28 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
	w[12] = w[12] << 1 | w[12] >> 31;

	//29 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
	w[13] = w[13] << 1 | w[13] >> 31;

	//30 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
	w[14] = w[14] << 1 | w[14] >> 31;

	//31 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
	w[15] = w[15] << 1 | w[15] >> 31;

	//32 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
	w[0] = w[0] << 1 | w[0] >> 31;

	//33 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
	w[1] = w[1] << 1 | w[1] >> 31;

	//34 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
	w[2] = w[2] << 1 | w[2] >> 31;

	//35 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
	w[3] = w[3] << 1 | w[3] >> 31;

	//36 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
	w[4] = w[4] << 1 | w[4] >> 31;

	//37 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
	w[5] = w[5] << 1 | w[5] >> 31;

	//38 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
	w[6] = w[6] << 1 | w[6] >> 31;

	//39 of 20-40
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
	w[7] = w[7] << 1 | w[7] >> 31;

	k = 0x8F1BBCDC;
	//40 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
	w[8] = w[8] << 1 | w[8] >> 31;

	//41 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
	w[9] = w[9] << 1 | w[9] >> 31;

	//42 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
	w[10] = w[10] << 1 | w[10] >> 31;

	//43 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
	w[11] = w[11] << 1 | w[11] >> 31;

	//44 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
	w[12] = w[12] << 1 | w[12] >> 31;

	//45 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
	w[13] = w[13] << 1 | w[13] >> 31;

	//46 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
	w[14] = w[14] << 1 | w[14] >> 31;

	//47 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
	w[15] = w[15] << 1 | w[15] >> 31;

	//48 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
	w[0] = w[0] << 1 | w[0] >> 31;

	//49 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
	w[1] = w[1] << 1 | w[1] >> 31;

	//50 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
	w[2] = w[2] << 1 | w[2] >> 31;

	//51 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
	w[3] = w[3] << 1 | w[3] >> 31;

	//52 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
	w[4] = w[4] << 1 | w[4] >> 31;

	//53 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
	w[5] = w[5] << 1 | w[5] >> 31;

	//54 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
	w[6] = w[6] << 1 | w[6] >> 31;

	//55 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
	w[7] = w[7] << 1 | w[7] >> 31;

	//56 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
	w[8] = w[8] << 1 | w[8] >> 31;

	//57 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
	w[9] = w[9] << 1 | w[9] >> 31;

	//58 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
	w[10] = w[10] << 1 | w[10] >> 31;

	//59 of 40-60
	f = (b & c) | (b & d) | (c & d);
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
	w[11] = w[11] << 1 | w[11] >> 31;

	k = 0xCA62C1D6;

	//60 of 60-64
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
	w[12] = w[12] << 1 | w[12] >> 31;

	//61 of 60-64
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
	w[13] = w[13] << 1 | w[13] >> 31;

	//62 of 60-64
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
	w[14] = w[14] << 1 | w[14] >> 31;

	//63 of 60-64
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
	w[15] = w[15] << 1 | w[15] >> 31;


	//64 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//65 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//66 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//67 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//68 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//69 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//70 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//71 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//72 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//73 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//74 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//75 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//76 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//77 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//78 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;

	//79 of 64-80
	f = b ^ c ^ d;
	temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
	e = d;
	d = c;
	c = (b << 30) | (b >> 2);
	b = a;
	a = temp;



	h.h1 += a;
	h.h2 += b;
	h.h3 += c;
	h.h4 += d;
	h.h5 += e;

}


__device__ void xorpads(uint32_t *pad, uint32_t* key)
{
	#pragma unroll 16
	for (unsigned i = 0; i < 16; i++)
		*(pad + i) = *(pad + i) ^ *(key + i);
}

uint32_t opad[16] = {0x5c5c5c5c,0x5c5c5c5c,0x5c5c5c5c,0x5c5c5c5c,
		     0x5c5c5c5c,0x5c5c5c5c,0x5c5c5c5c,0x5c5c5c5c,
		     0x5c5c5c5c,0x5c5c5c5c,0x5c5c5c5c,0x5c5c5c5c,
		     0x5c5c5c5c,0x5c5c5c5c,0x5c5c5c5c,0x5c5c5c5c,};
uint32_t ipad[16] = {0x36363636,0x36363636,0x36363636,0x36363636,
		     0x36363636,0x36363636,0x36363636,0x36363636,
		     0x36363636,0x36363636,0x36363636,0x36363636,
		     0x36363636,0x36363636,0x36363636,0x36363636,};

__global__ void computeHMAC_SHA1(char* buf, char* keys,  uint32_t *offsets, uint16_t *lengths, uint32_t *outputs, int N, uint8_t * checkbits)
{
	uint32_t w_register[16];

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N) {
		uint32_t *w = w_register;
		hash_digest_t h;
		uint32_t offset = offsets[index];
		uint16_t length = lengths[index];
		uint32_t *out = outputs + 5 * index;


		for (unsigned i = 0; i < 16; i++)
			w[i] = 0x36363636;
		xorpads(w, (uint32_t*)(keys + 64 * index));


		h.h1 = 0x67452301;
		h.h2 = 0xEFCDAB89;
		h.h3 = 0x98BADCFE;
		h.h4 = 0x10325476;
		h.h5 = 0xC3D2E1F0;

		//SHA1 compute on ipad
		computeSHA1Block((char*)w, w, 0, 64, h);

		//SHA1 compute on mesage
		unsigned num_iter = (length + 63 + 9) / 64;
		for (unsigned i = 0; i < num_iter; i++)
			computeSHA1Block(buf + offset , w, i * 64  , length , h);

		*(out)   = swap(h.h1);
		*(out+1) = swap(h.h2);
		*(out+2) = swap(h.h3);
		*(out+3) = swap(h.h4);
		*(out+4) = swap(h.h5);

		h.h1 = 0x67452301;
		h.h2 = 0xEFCDAB89;
		h.h3 = 0x98BADCFE;
		h.h4 = 0x10325476;
		h.h5 = 0xC3D2E1F0;

		for (unsigned i = 0; i < 16; i++)
			w[i] = 0x5c5c5c5c;

		xorpads(w, (uint32_t*)(keys + 64 * index));

		//SHA 1 compute on opads
		computeSHA1Block((char*)w, w, 0, 64, h);

		//SHA 1 compute on (hash of ipad|m)
		computeSHA1Block((char*)out, w, 0, 20, h);

		*(out)   = swap(h.h1);
		*(out+1) = swap(h.h2);
		*(out+2) = swap(h.h3);
		*(out+3) = swap(h.h4);
		*(out+4) = swap(h.h5);
	}
        __syncthreads();

	if (threadIdx.x == 0)
		*(checkbits + blockIdx.x) = 1;

}

void hmac_sha1_gpu(char* buf, char* keys,  uint32_t *offsets, uint16_t *lengths,
		   uint32_t *outputs, int N, uint8_t * checkbits,
		   unsigned threads_per_blk, hipStream_t stream)
{
	int num_blks = (N + threads_per_blk - 1) / threads_per_blk;
	if (stream == 0) {
		computeHMAC_SHA1<<<num_blks, threads_per_blk>>>(
		       buf, keys, offsets, lengths, outputs, N, checkbits);
	} else  {
		computeHMAC_SHA1<<<num_blks, threads_per_blk, 0, stream>>>(
		       buf, keys, offsets, lengths, outputs, N, checkbits);
	}
}

